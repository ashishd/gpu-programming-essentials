
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "ERROR %d (%s:%d): %s\n", (X), __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})

#define NTHREADS 1024 
#define ALL_THREADS_MASK 0xffffffff
#define WARPSIZE 32


__global__ void vector_reduction_kernel(unsigned char *values, unsigned int nitems, unsigned long long* result){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ unsigned int partial_sums[WARPSIZE];
    unsigned int warpId = threadIdx.x / warpSize;
    unsigned int laneId = threadIdx.x % warpSize; 
    unsigned int gridSize = gridDim.x * blockDim.x;
    unsigned int nloops = (nitems + gridSize  - 1) / gridSize;
    unsigned int l = 0;
    unsigned int myvalue;

    while(l++ < nloops){
        if(idx < nitems){ 
            myvalue = values[idx]; 
        }else{
            myvalue = 0;
        }
 
        // step 1
        for(unsigned int i = warpSize/2; i >= 1; i >>= 1){
            unsigned int up = __shfl_down_sync(ALL_THREADS_MASK, myvalue, i, warpSize); 
            if(laneId < i){
                myvalue += up; 
            }
        }
        if(laneId == 0 && warpId > 0) partial_sums[warpId] = myvalue;
       
        __syncthreads();
        // step 2
        if(warpId == 0){
            if(laneId > 0) myvalue = partial_sums[laneId];
            for(unsigned int i = warpSize/2; i >= 1; i >>= 1){
                unsigned int up = __shfl_down_sync(ALL_THREADS_MASK, myvalue, i, warpSize); 
                    if(laneId < i){
                        myvalue += up; 
                    }
                }
            if(laneId == 0) atomicAdd(result, myvalue);
        }  
        idx += gridSize;
        __syncthreads();
    }
}



int main(int argc, char **argv){
    
    unsigned int nitems = 1e9; 
    unsigned char *values = (unsigned char*) malloc(sizeof(unsigned int) * nitems);
    if(!values){
        fprintf(stderr, "Error while allocating memory\n");
        return -1;
    }
    // Initialise the vector of n elements to random values
    unsigned long long correct_result = 0;
    for(int i = 0; i < nitems; i++){
        values[i] = (i + 1) % 128;
        correct_result += values[i];
    }
    unsigned long long sum = 0ull;
    unsigned long long *dev_sum;
    unsigned char *dev_values;
    CUDA_CHECK_ERROR(hipMalloc(&dev_values, sizeof(unsigned char) * nitems));
    CUDA_CHECK_ERROR(hipMalloc(&dev_sum, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemset(dev_sum, 0, sizeof(unsigned long long)));
    CUDA_CHECK_ERROR(hipMemcpy(dev_values, values, sizeof(unsigned char) * nitems, hipMemcpyHostToDevice));
    struct hipDeviceProp_t props;
    CUDA_CHECK_ERROR(hipGetDeviceProperties(&props, 0));
    unsigned int nblocks = props.multiProcessorCount * 2;
    printf("Number of cuda blocks: %u\n", nblocks);
    hipEvent_t start, stop;
    CUDA_CHECK_ERROR(hipEventCreate(&start));
    CUDA_CHECK_ERROR(hipEventCreate(&stop));
    CUDA_CHECK_ERROR(hipEventRecord(start)); 
    vector_reduction_kernel<<<nblocks, NTHREADS>>>(dev_values, nitems, dev_sum);
    CUDA_CHECK_ERROR(hipGetLastError());
    CUDA_CHECK_ERROR(hipEventRecord(stop)); 
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    CUDA_CHECK_ERROR(hipMemcpy(&sum, dev_sum, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    float time_spent;
    CUDA_CHECK_ERROR(hipEventElapsedTime(&time_spent, start, stop));
    printf("Result: %llu - Time elapsed: %f\n", sum, time_spent/1000.0f);
    if(correct_result != sum) {
        fprintf(stderr, "Error: sum is not correct, should be %llu\n", correct_result);
        return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;

}